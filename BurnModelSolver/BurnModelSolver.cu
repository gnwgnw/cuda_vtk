#include "BurnModelSolver.hpp"
#include "Solver/cuda_utils.cuh"


BurnModelSolver::~BurnModelSolver() {
  if (d_params) {
    hipFree(d_params);
  }
}

void BurnModelSolver::set_params(const Params &new_params) {
  params = new_params;

  size_t param_size = sizeof(Params);

  if (d_params) {
    hipFree(d_params);
  }
  cuda_check_error(hipMalloc((void **) &d_params, param_size));
  cuda_check_error(hipMemcpy(d_params, &params, param_size, hipMemcpyHostToDevice));
}
