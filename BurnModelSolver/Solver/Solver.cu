#include "Solver.hpp"
#include "cuda_utils.cuh"


void Solver::step() {
  if (!done) {
    next_step();
    cuda_check_error(hipGetLastError());

    std::swap(d_y_in, d_y_out);

    t += tau;
    if (t > t_end) {
      done = true;
    }

    if (is_y_current) {
      is_y_current = false;
    }
  }
}

void Solver::alloc_mem() {
  y.resize(N);
  x.resize(N);
  cuda_check_error(hipMalloc((void **) &d_y_in, size));
  cuda_check_error(hipMalloc((void **) &d_y_out, size));
  cuda_check_error(hipMemcpy(d_y_in, y.data(), size, hipMemcpyHostToDevice));
  cuda_check_error(hipMemcpy(d_y_out, y.data(), size, hipMemcpyHostToDevice));
}

void Solver::free_mem() {
  cuda_check_error(hipFree(d_y_in));
  cuda_check_error(hipFree(d_y_out));
}

void Solver::copy_from_device() {
  cuda_check_error(hipMemcpy(y.data(), d_y_in, size, hipMemcpyDeviceToHost));
  is_y_current = true;
}
